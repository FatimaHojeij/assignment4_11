#include "hip/hip_runtime.h"

#include "common.h"
#include "timer.h"

#define IN_TILE_DIM 32
#define OUT_TILE_DIM ((IN_TILE_DIM) - 2*(MASK_RADIUS))

__constant__ float mask_c[MASK_DIM][MASK_DIM];

__global__ void convolution_tiled_kernel(float* input, float* output, unsigned int width, unsigned int height) {

	__shared__ float input_s[IN_TILE_DIM][IN_TILE_DIM];
    // TODO
	int outRow = blockIdx.y*OUT_TILE_DIM + threadIdx.y;
	int outCol = blockIdx.x*OUT_TILE_DIM + threadIdx.x;
	int inRow = blockIdx.y*blockDim.y + threadIdx.y;
	int inCol = blockIdx.x*blockDim.x + threadIdx.x;
	if (inRow < height && inRow >= 0 && inCol < width && inCol >= 0) {
		input_s[threadIdx.y][threadIdx.x] = input[inRow*width + inCol];
	}
	__syncthreads();
	
	if(inRow<OUT_TILE_DIM && inCol<OUT_TILE_DIM ){
		float sum = 0.0f;
        for(int maskRow = 0; maskRow < MASK_DIM; ++maskRow) {
            for(int maskCol = 0; maskCol < MASK_DIM; ++maskCol) {
                int inRow = outRow - MASK_RADIUS + maskRow;
                int inCol = outCol - MASK_RADIUS + maskCol;
                if(inRow >= 0 && inRow < height && inCol >= 0 && inCol < width) {
                    sum += mask_c_[maskRow][maskCol]*input_s[inRow*width + inCol];
                }
            }
			__syncthreads();
        }
        output[outRow*width + outCol] = sum;
	}
}

void copyMaskToGPU(float mask[][MASK_DIM]) {

    // Copy mask to constant memory

    // TODO
	hipMemcpyToSymbol(HIP_SYMBOL(mask_c), mask, MASK_DIM*MASK_DIM*sizeof(float));

}

void convolution_tiled_gpu(float* input_d, float* output_d, unsigned int width, unsigned int height) {

    // Call kernel

    // TODO
	dim3 numThreadsPerBlock(IN_TILE_DIM, IN_TILE_DIM);
	dim3 numBlocks((width + OUT_TILE_DIM - 1)/OUT_TILE_DIM, (height + OUT_TILE_DIM - 1)/OUT_TILE_DIM);
	convolution_tiled_kernel <<< numBlocks, numThreadsPerBlock >>> (input_d, output_d, width, height);


}

